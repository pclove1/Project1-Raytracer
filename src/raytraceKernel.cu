#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include <cutil_math.h>
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
  ray r;
  r.origin = eye;

  glm::vec3 right = glm::normalize(glm::cross(view, up));
  //TODO: use glm::tan and glm::radians
  float d = 0.5f * resolution.y / tan(fov.y*(PI/180.f)); // distance from the eye to the image plane
  r.direction = glm::normalize(view*d + right*(0.5f*resolution.x - x) + up*(0.5f*resolution.y - y));
  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors, 
                            staticGeom* geoms, int numberOfGeoms,
							const material* materials, int numOfMaterials,
							const int* lightIndices, int numOfLights){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	if((x<=resolution.x && y<=resolution.y)){
		// colors[index] = generateRandomNumberFromThread(resolution, time, x, y);
		colors[index] = glm::vec3(0.f, 0.f, 0.f);
		ray r = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov);
		
		glm::vec3 intersectionPoint, normal;
		float intersectionDistance;
		int intersectionGeomInd = findClosestIntersection(geoms, numberOfGeoms, r,
			&intersectionPoint, &normal, &intersectionDistance);

		if (intersectionGeomInd != -1) { // found the closest front object
			const material& objectMaterial = materials[geoms[intersectionGeomInd].materialid];
			glm::vec3 diffuseColor = objectMaterial.color;
			glm::vec3 specularColor = objectMaterial.specularColor;

			if (objectMaterial.emittance > EPSILON) {
				// object to be rendered is a light source
				colors[index] = diffuseColor;
				return;
			}

			/* Phong Illumination Model */
			/* ka*diffuse_color + kd*diffuse_color*(N*L) + ks*specular_color*(N*H)^exp_n 
			 * N(normal) : unit vector, the direction of the surface normal at the intersection 
			 * L(ligtDirection) : unit vector, the direction of the vector to the light source from the intersection 
			 * H : unit vector, the direction that is halfway between the direction to the light and the direction to the viewer */
			glm::vec3 diffuse_sum(0.f, 0.f, 0.f);
			glm::vec3 specular_sum(0.f, 0.f, 0.f);
			for (int i = 0; i < numOfLights; i++) { // for each light source
				int lightInd = lightIndices[i];
				const staticGeom& light = geoms[lightInd];
				glm::vec3 lightCenter = multiplyMV(light.transform, glm::vec4(0.f, 0.f, 0.f, 1.0f));
				glm::vec3 lightDirection = glm::normalize(lightCenter - intersectionPoint);

				// check occulusion for shadow
				// NOTE: move the intersection point toward each light a little bit to avoid numerical error
				ray lightRay; lightRay.origin = intersectionPoint + lightDirection*float(RAY_BIAS_AMOUNT); lightRay.direction = lightDirection;
				int obstacleGeomInd = findClosestIntersection(geoms, numberOfGeoms, lightRay);
				if (obstacleGeomInd != lightInd) {
					continue;
				}
				
				glm::vec3 V = glm::normalize(-r.direction);
				glm::vec3 H = glm::normalize(lightDirection + V);

				const material& lightMaterial = materials[light.materialid];
				glm::vec3 lightColor = lightMaterial.color;
				diffuse_sum += lightColor * max(0.f, glm::dot(normal, lightDirection));

				if (glm::dot(normal, H) > EPSILON) {
					specular_sum += lightColor * (glm::pow(glm::dot(normal, H), objectMaterial.specularExponent));
				}
			}

			colors[index] = glm::clamp(0.3f*diffuseColor + diffuseColor*diffuse_sum + specularColor*specular_sum, 0.f, 1.f); 
			//colors[index] = glm::clamp(0.3f*diffuseColor + diffuseColor*diffuse_sum, 0.f, 1.f); 
		}
	}
}


//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  //package geometry and indices for lights and send them to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  int* lightIndices = new int[numberOfGeoms];
  int numOfLights = 0;
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;

	if (materials[newStaticGeom.materialid].emittance > EPSILON) {
		lightIndices[numOfLights++] = i;
	}
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  int* cudaLightIndices = NULL;
  hipMalloc((void**)&cudaLightIndices, numOfLights*sizeof(int));
  hipMemcpy( cudaLightIndices, lightIndices, numOfLights*sizeof(int), hipMemcpyHostToDevice);

  // package materials and send to GPU
  material* cudaMaterials = NULL;
  hipMalloc((void**)&cudaMaterials, numberOfMaterials*sizeof(material));
  hipMemcpy(cudaMaterials, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);
  
  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

  //kernel launches
  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, 
	  cudageoms, numberOfGeoms, cudaMaterials, numberOfMaterials, cudaLightIndices, numOfLights);

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  hipFree( cudaMaterials );
  hipFree( cudaLightIndices );
  delete geomList;
  delete lightIndices;

  // make certain the kernel has completed 
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}
